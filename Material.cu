#include "hip/hip_runtime.h"
#include "Material.h"
#define PI 3.14159265359f
__device__ bool MultiMaterial::scatter(
	const Ray& r_in, const HitRecord& rec, Vec3& attenuation, Ray& scattered, hiprandState* local_rand_state) const {

	Vec3 normal = rec.normal;
	if (textures[textureTypeNormal] != -1) {
		float4 col = tex2D<float4>((hipTextureObject_t)textures[textureTypeNormal], rec.uvCoords.x, rec.uvCoords.y);
		normal = Vec3(col.x, col.y, col.z);
		normal = 2 * normal - 1;
		Vec3 a = Vec3(rec.tangent.x, rec.bitangent.x, rec.normal.x);
		Vec3 b = Vec3(rec.tangent.y, rec.bitangent.y, rec.normal.y);
		Vec3 c = Vec3(rec.tangent.z, rec.bitangent.z, rec.normal.z);
		normal = d_normalize(Vec3(dot(a, normal), dot(b, normal), dot(c, normal)));
	}

	if (textures[textureTypeDiffuse] == -1)
		attenuation = colors[textureTypeDiffuse];
	else
	{
		float4 col = tex2D<float4>((hipTextureObject_t)textures[textureTypeDiffuse], rec.uvCoords.x, rec.uvCoords.y);
		//printf("%d\n", cudaTextures[textureTypeDiffuse].texObject);
		attenuation = Vec3(col.x, col.y, col.z);
	}

	float specular;
	if (textures[textureTypeSpecular] == -1)
		specular = colors[textureTypeSpecular].x;
	else
	{
		float4 col = tex2D<float4>((hipTextureObject_t)textures[textureTypeSpecular], rec.uvCoords.x, rec.uvCoords.y);
		//printf("%d\n", cudaTextures[textureTypeDiffuse].texObject);
		specular = col.x;
	}
	float roughness;
	if (textures[textureTypeRoughness] == -1)
		roughness = colors[textureTypeRoughness].x;
	else
	{
		float4 col = tex2D<float4>((hipTextureObject_t)textures[textureTypeRoughness], rec.uvCoords.x, rec.uvCoords.y);
		//printf("%d\n", cudaTextures[textureTypeDiffuse].texObject);
		roughness = col.x;
	}

	if(hiprand_uniform(local_rand_state) < specular)
	{
		Vec3 reflected = reflect(d_normalize(r_in.direction), normal);
		scattered = Ray(rec.p, reflected + roughness * random_unit_vector(local_rand_state));
		//attenuation *= dot(scattered.direction, normal);
		return dot(scattered.direction, normal) > 0;
	}
	else {
		Vec3 dir = random_on_hemisphere(normal, local_rand_state);
		// Catch degenerate scatter direction
		if (dir.near_zero())
			dir = normal;
		auto cos_theta = dot(normal, d_normalize(dir));
		//attenuation *= cos_theta < 0 ? 0 : cos_theta / PI;
		scattered = Ray(rec.p, dir);
		return true;
	}	
}