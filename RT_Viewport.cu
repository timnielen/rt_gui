#include "hip/hip_runtime.h"
#include "RT_Viewport.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include "Sphere.h"

#include "raytracing.h"

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
	if (result) {
		std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
			file << ":" << line << " '" << func << "' \n";
		// Make sure we call CUDA Device Reset before exiting
		hipDeviceReset();
		exit(99);
	}
}

__global__
void add(int n, float* x, float* y)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n) return;
	y[i] = x[i] + y[i];
}


RT_Viewport::RT_Viewport() : size({ -1,-1 }) {
	unsigned int dCount = 0;
	int devices[10];
	hipDeviceProp_t prop;
	checkCudaErrors(hipGLGetDevices(&dCount, devices, 10, hipGLDeviceListAll));
	std::cout << "Devices used to render image (OpenGL):" << std::endl;
	for (int i = 0; i < dCount; i++) {
		hipGetDeviceProperties(&prop, devices[i]);
		std::cout << devices[i] << "\t" << prop.name << std::endl;
	}
	int cudaDevice;
	hipGetDevice(&cudaDevice);
	hipGetDeviceProperties(&prop, cudaDevice);
	std::cout << "Cuda device:" << std::endl;
	std::cout << cudaDevice << "\t" << prop.name << std::endl;
	//checkCudaErrors(hipSetDevice(devices[0]));


	glGenTextures(1, &texture);
	checkCudaErrors(hipMallocManaged((void**)&cam, sizeof(RT_Camera)));

	checkCudaErrors(hipMalloc((void**)&objects, 4*sizeof(Hitable*)));
	checkCudaErrors(hipMalloc((void**)&scene, sizeof(Hitable*)));
	init_scene<<<1, 1 >>>(scene, objects);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	
}


float4* data = nullptr;
//returns wether to the current framebuffer is drawable
bool RT_Viewport::updateFramebuffer() {
	ImVec2 vMin = ImGui::GetWindowContentRegionMin();
	ImVec2 vMax = ImGui::GetWindowContentRegionMax();
	ImVec2 newViewportSize = ImVec2(vMax.x - vMin.x, vMax.y - vMin.y);
	if (size.x == newViewportSize.x && size.y == newViewportSize.y)
		return resizeFinished;

	//is currently resizing? -> not drawable
	if (ImGui::IsMouseDown(ImGuiMouseButton_Left))
		return false;

	resizeFinished = false;
	size = newViewportSize;

	//is window hidden? -> not drawable
	if (size.x == 0 || size.y == 0)
		return false;

	cam->update(size.x, size.y);
	std::cout << "new texture: " << size.x << " " << size.y << std::endl;
	glDeleteTextures(1, &texture);
	glGenTextures(1, &texture);

	if (data != nullptr) {
		hipFree(data); 
	}
	hipMallocManaged(&data, sizeof(float4) * size.x * size.y);
	for (int i = 0; i < size.x * size.y; i++)
		data[i] = make_float4(0.1f, 0.5f, 1.0f, 1.0f);

	glBindTexture(GL_TEXTURE_2D, texture);
	{
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
		glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA32F, size.x, size.y, 0, GL_RGBA, GL_FLOAT, data);
	}
	glBindTexture(GL_TEXTURE_2D, 0);

	renderedImage.init(texture);

	//init randoms
	if(d_rand_state != nullptr)
		checkCudaErrors(hipFree(d_rand_state));
	checkCudaErrors(hipMalloc((void**)&d_rand_state, size.x * size.y * sizeof(hiprandState)));

	dim3 blocks(size.x / blockW + 1, size.y / blockH + 1);
	dim3 threads(blockW, blockH);
	render_init << <blocks, threads >> > (size.x, size.y, d_rand_state);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	accumulation = 0;

	resizeFinished = true;
	return true;
}

bool first = true;
void RT_Viewport::render(float deltaTime) {
	ImGui::Begin("RT Viewport");
	bool fbDrawable = updateFramebuffer();
	if (size.x <= 0 || size.y <= 0) {
		ImGui::End();
		return;
	}
	if (fbDrawable && !first) {
		invokeRenderProcedure();
	}
	first = false;

	ImGui::Image((void*)texture, size); // , { 0, 1 }, { 1, 0 });
	ImGui::End();
}


void RT_Viewport::invokeRenderProcedure() {
	// Render our buffer
	dim3 blocks(size.x / blockW + 1, size.y / blockH + 1);
	dim3 threads(blockW, blockH);

	accumulation++;
	renderedImage.map();
	render_image<<<blocks, threads>>> (renderedImage.getSurface(), size.x, size.y, cam, scene, d_rand_state, samples, max_steps, accumulation);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	renderedImage.unmap();

	checkCudaErrors(hipStreamSynchronize(0));
}

RT_Viewport::~RT_Viewport()
{
	hipFree(data);
	renderedImage.destroy();
	checkCudaErrors(hipDeviceSynchronize());
	free_scene<<<1, 1 >>>(scene, objects, 4);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipFree(objects));
	checkCudaErrors(hipFree(scene));
	checkCudaErrors(hipFree(cam));
	glDeleteTextures(1, &texture);
}

void ImageResource::init(unsigned int texture) {
	if (gfxRes != NULL)
		destroy();
	auto e = hipGraphicsGLRegisterImage(&gfxRes, texture, GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore);
	checkCudaErrors(e);
}

void ImageResource::map() {

	checkCudaErrors(hipGraphicsMapResources(1, &gfxRes));

	hipArray_t viewCudaArray;
	checkCudaErrors(hipGraphicsSubResourceGetMappedArray(&viewCudaArray, gfxRes, 0, 0));
	hipResourceDesc viewCudaArrayResourceDesc;
	{
		viewCudaArrayResourceDesc.resType = hipResourceTypeArray;
		viewCudaArrayResourceDesc.res.array.array = viewCudaArray;
	}
	checkCudaErrors(hipCreateSurfaceObject(&viewCudaSurfaceObject, &viewCudaArrayResourceDesc));
}

void ImageResource::unmap() {
	checkCudaErrors(hipGraphicsUnmapResources(1, &gfxRes));
}

void ImageResource::destroy() {
	checkCudaErrors(hipGraphicsUnregisterResource(gfxRes));
	gfxRes = NULL;
	checkCudaErrors(hipDestroySurfaceObject(viewCudaSurfaceObject));
}