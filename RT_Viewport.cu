#include "hip/hip_runtime.h"
#include "RT_Viewport.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include "Sphere.h"
#include "File.h"
#include "raytracing.h"
#include "cuda_helper.h"

void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
	if (result) {
		std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
			file << ":" << line << " '" << func << "' \n";
		// Make sure we call CUDA Device Reset before exiting
		hipDeviceReset();
		exit(99);
	}
}

__global__
void add(int n, float* x, float* y)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n) return;
	y[i] = x[i] + y[i];
}


RT_Viewport::RT_Viewport() : size({ -1,-1 }) {
	unsigned int dCount = 0;
	int devices[10];
	hipDeviceProp_t prop;
	checkCudaErrors(hipGLGetDevices(&dCount, devices, 10, hipGLDeviceListAll));
	std::cout << "Devices used to render image (OpenGL):" << std::endl;
	for (int i = 0; i < dCount; i++) {
		hipGetDeviceProperties(&prop, devices[i]);
		std::cout << devices[i] << "\t" << prop.name << std::endl;
	}
	int cudaDevice;
	hipGetDevice(&cudaDevice);
	hipGetDeviceProperties(&prop, cudaDevice);
	std::cout << "Cuda device:" << std::endl;
	std::cout << cudaDevice << "\t" << prop.name << std::endl;
	//checkCudaErrors(hipSetDevice(devices[0]));


	glGenTextures(1, &texture);
	checkCudaErrors(hipMallocManaged((void**)&camera, sizeof(Camera)));
	*camera = Camera();
	camera->setPosition(glm::vec3(0, 0, 3));

	//checkCudaErrors(hipMallocManaged((void**)&objects, 4*sizeof(Hitable*)));
	//checkCudaErrors(hipMallocManaged((void**)&scene, sizeof(Hitable*)));
	//d_Model model = d_Model(Model("assets/Survival_BackPack_2/backpack.obj"));
	d_Model model = d_Model(Model("assets/cube.obj"));



	scene = model.hitable;
	/*init_scene<<<1, 1 >>>(scene, model.hitables);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());*/

	hdri.init(load_texture("./assets/hdri/rural_crossroads_4k.hdr"));

}


float4* data = nullptr;
//returns wether to the current framebuffer is drawable
bool RT_Viewport::updateFramebuffer() {
	ImVec2 vMin = ImGui::GetWindowContentRegionMin();
	ImVec2 vMax = ImGui::GetWindowContentRegionMax();
	ImVec2 newViewportSize = ImVec2(vMax.x - vMin.x, vMax.y - vMin.y);
	if (size.x == newViewportSize.x && size.y == newViewportSize.y)
		return resizeFinished;

	//is currently resizing? -> not drawable
	if (ImGui::IsMouseDown(ImGuiMouseButton_Left))
		return false;

	resizeFinished = false;
	size = newViewportSize;

	//is window hidden? -> not drawable
	if (size.x == 0 || size.y == 0)
		return false;

	//cam->updateViewportSize(size.x, size.y);
	camera->setProjection(45.0f, size.x, size.y, 0.1f, 1000.0f);
	camera->updateView();

	std::cout << "new texture: " << size.x << " " << size.y << std::endl;
	glDeleteTextures(1, &texture);
	glGenTextures(1, &texture);

	if (data != nullptr) {
		hipFree(data); 
	}
	hipMallocManaged(&data, sizeof(float4) * size.x * size.y);
	for (int i = 0; i < size.x * size.y; i++)
		data[i] = make_float4(0.1f, 0.5f, 1.0f, 1.0f);

	glBindTexture(GL_TEXTURE_2D, texture);
	{
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
		glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA32F, size.x, size.y, 0, GL_RGBA, GL_FLOAT, data);
	}
	glBindTexture(GL_TEXTURE_2D, 0);

	renderedImage.init(texture);

	//init randoms
	if(d_rand_state != nullptr)
		checkCudaErrors(hipFree(d_rand_state));
	checkCudaErrors(hipMalloc((void**)&d_rand_state, size.x * size.y * sizeof(hiprandState)));

	dim3 blocks(size.x / blockW + 1, size.y / blockH + 1);
	dim3 threads(blockW, blockH);
	render_init << <blocks, threads >> > (size.x, size.y, d_rand_state);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	accumulation = 0;

	resizeFinished = true;
	return true;
}

bool first = true;
void RT_Viewport::render(float deltaTime) {
	ImGui::Begin("RT Viewport");
	bool fbDrawable = updateFramebuffer();
	if (size.x <= 0 || size.y <= 0) {
		ImGui::End();
		return;
	}

	//update camera
	if (ImGui::IsWindowFocused()) {
		if (ImGui::IsMouseDown(ImGuiMouseButton_Left)) {
			ImVec2 mousePos = ImGui::GetMousePos();
			if (firstMouse) {
				firstMouse = false;
				lastMousePos = mousePos;
			}
			else {
				float sensitivity = 0.1f;
				ImVec2 offset = { sensitivity * (mousePos.x - lastMousePos.x), sensitivity * (lastMousePos.y - mousePos.y) };
				lastMousePos = mousePos;

				camera->yaw += offset.x;
				camera->pitch += offset.y;

				if (camera->pitch > 89.0f)
					camera->pitch = 89.0f;
				if (camera->pitch < -89.0f)
					camera->pitch = -89.0f;

				camera->updateView();
				accumulation = 0;
			}
		}
		else
			firstMouse = true;

		float cameraSpeed = 2.0f;
		cameraSpeed *= deltaTime;
		//std::cout << deltaTime << std::endl;
		glm::vec3 moveDir = glm::vec3(0);
		if (ImGui::IsKeyDown(ImGuiKey_W))
			moveDir += camera->direction;
		if (ImGui::IsKeyDown(ImGuiKey_A))
			moveDir += camera->right;
		if (ImGui::IsKeyDown(ImGuiKey_S))
			moveDir -= camera->direction;
		if (ImGui::IsKeyDown(ImGuiKey_D))
			moveDir -= camera->right;

		if (moveDir != glm::vec3(0)) {
			moveDir = glm::normalize(moveDir) * cameraSpeed;
			camera->position += moveDir;
			camera->updateView();
			accumulation = 0;
		}
	}
	else
		firstMouse = true;

	if (fbDrawable && !first) {
		invokeRenderProcedure();
	}
	first = false;

	ImGui::Image((void*)texture, size);// , { 0, 1 }, { 1, 0 });
	ImGui::End();
}


void RT_Viewport::invokeRenderProcedure() {
	// Render our buffer
	dim3 blocks(size.x / blockW + 1, size.y / blockH + 1);
	dim3 threads(blockW, blockH);

	accumulation++;
	hdri.mapTexture();
	renderedImage.mapSurface();
	render_image<<<blocks, threads>>> (renderedImage.getSurface(), size.x, size.y, camera, scene, hdri.getTexture(), d_rand_state, samples, max_steps, accumulation);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	renderedImage.unmap();
	hdri.unmap();

	//checkCudaErrors(hipStreamSynchronize(0));
}

RT_Viewport::~RT_Viewport()
{
	hipFree(data);
	renderedImage.destroy();
	hdri.destroy();
	checkCudaErrors(hipDeviceSynchronize());
	free_scene<<<1, 1 >>>(scene, objects, 4);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipFree(objects));
	checkCudaErrors(hipFree(scene));
	checkCudaErrors(hipFree(camera));
	glDeleteTextures(1, &texture);
	glDeleteTextures(1, &hdri.texture);
}

void ImageResource::init(unsigned int tex, int flags) {
	texture = tex;
	if (gfxRes != NULL)
		destroy();
	auto e = hipGraphicsGLRegisterImage(&gfxRes, texture, GL_TEXTURE_2D, flags);
	checkCudaErrors(e);
}

void ImageResource::mapSurface() {

	checkCudaErrors(hipGraphicsMapResources(1, &gfxRes));

	hipArray_t viewCudaArray;
	checkCudaErrors(hipGraphicsSubResourceGetMappedArray(&viewCudaArray, gfxRes, 0, 0));
	hipResourceDesc viewCudaArrayResourceDesc;
	{
		viewCudaArrayResourceDesc.resType = hipResourceTypeArray;
		viewCudaArrayResourceDesc.res.array.array = viewCudaArray;
	}
	checkCudaErrors(hipCreateSurfaceObject(&viewCudaSurfaceObject, &viewCudaArrayResourceDesc));
}

void ImageResource::mapTexture() {

	checkCudaErrors(hipGraphicsMapResources(1, &gfxRes));

	struct hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.addressMode[0] = hipAddressModeWrap;
	texDesc.addressMode[1] = hipAddressModeWrap;
	texDesc.filterMode = hipFilterModePoint;
	texDesc.readMode = hipReadModeElementType;
	texDesc.normalizedCoords = 1;

	hipArray_t viewCudaArray;
	checkCudaErrors(hipGraphicsSubResourceGetMappedArray(&viewCudaArray, gfxRes, 0, 0));
	hipResourceDesc viewCudaArrayResourceDesc;
	{
		viewCudaArrayResourceDesc.resType = hipResourceTypeArray;
		viewCudaArrayResourceDesc.res.array.array = viewCudaArray;
	}
	checkCudaErrors(hipCreateTextureObject(&viewCudaTextureObject, &viewCudaArrayResourceDesc, &texDesc, NULL));
}

void ImageResource::unmap() {
	checkCudaErrors(hipGraphicsUnmapResources(1, &gfxRes));
}

void ImageResource::destroy() {
	checkCudaErrors(hipGraphicsUnregisterResource(gfxRes));
	gfxRes = NULL;
	checkCudaErrors(hipDestroySurfaceObject(viewCudaSurfaceObject));
}